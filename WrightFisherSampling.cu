#include "hip/hip_runtime.h"
/******************************************************************************************
**Code to generate samples from a Wright-Fisher diffusion with selection
**For each set of parameters, repeatedly proposes N values until total of SAMPSIZE accepted
**Set T, theta, range of s, and range of p at start of main program
**Set name of output file at end of main program
**Output file contains SAMPSIZE draws for each combination of s and p  
**Compile with...
/usr/local/cuda/bin/nvcc -arch sm_20 -L /usr/local/cuda/lib64 -lcurand -o wf WrightFisherSampling.cu `pkg-config --cflags --libs gsl`
**Execute with...   ./wf &
******************************************************************************************/

#include <stddef.h>  // NULL, size_t 
#include <math.h> // expf 
#include <stdio.h> // printf
#include <time.h> // time 
#include <assert.h> 
#include <gsl/gsl_randist.h>

#include <hiprand.h> 
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand_kernel.h> 

#include <iostream> 
#include <fstream> 

using namespace std; 

#define N 25000
#define SAMPSIZE 10000

double phiplus(double s, double theta)
{
    double phimax;
    if (0<=0.5/s*(s-2.0*theta) && 0.5/s*(s-2.0*theta)<=1)
        phimax = 1.0/8.0*pow(s-2.0*theta,2)+theta*s/2;
    else if (0.5/s*(s-2*theta)<0)
        phimax=theta*s/2;
    else
        phimax=-theta*s/2;
    return phimax;
}

void quick_sort(double* arr,int* ord,int low,int high)

{

 int pivot,j,i,temp2;

 double temp;
 if(low<high)

 {
  
  pivot = low;

  i = low;

  j = high;


 
 while(i<j)

  {

   while((arr[i]<=arr[pivot])&&(i<high))

   {

    i++;

   }

 
  while(arr[j]>arr[pivot])

   {

    j--;

   }

 
  if(i<j)

   {
 
    temp=arr[i];
     arr[i]=arr[j];

     arr[j]=temp;

     temp2=ord[i];
     ord[i]=ord[j];

     ord[j]=temp2;

   }

  }
  temp=arr[pivot];

  arr[pivot]=arr[j];

  arr[j]=temp;

  temp2=ord[pivot];

  ord[pivot]=ord[j];

  ord[j]=temp2;
  quick_sort(arr,ord,low,j-1);

  quick_sort(arr,ord,j+1,high);

 }

}

__global__ void TimeDiffs(int *dev_csp, unsigned int *dev_poi, double *dev_parms, double *dev_tss, double *dev_tdiffs)
{
  int bid=blockIdx.x;
   int size, start, firstt;
   size=dev_poi[bid];
   firstt=dev_csp[bid];
   start=firstt+bid;
   if (size==0)
     dev_tdiffs[start]=dev_parms[2];
   else
   {
     dev_tdiffs[start]=dev_tss[firstt];
     for (int j=1; j<size; j++)
     {
       dev_tdiffs[start+j]=dev_tss[firstt+j]-dev_tss[firstt+j-1];
     }
     dev_tdiffs[start+size]=dev_parms[2]-dev_tss[firstt+size-1];
   }
}

__global__ void NormApproxParams(int *dev_csp, unsigned int *dev_poi, double *dev_tdiffs, double *dev_parms, double *dev_means, double *dev_vars)
{
  int bid=blockIdx.x;
   int size, start;
   size=dev_poi[bid]+1;
   start=dev_csp[bid]+bid;
   for (int j=0; j<size; j++)
   {
    double beta, eta;
    beta=0.5*(dev_parms[1]-1)*dev_tdiffs[start+j];
    eta=beta/(exp(beta)-1);
    dev_vars[start+j]=abs(2.0*eta/dev_tdiffs[start+j]*pow(eta+beta,2)*(1+eta/(eta+beta)-2.0*eta));
    dev_means[start+j]=2.0*eta/dev_tdiffs[start+j];
   }
}
   
__global__ void KeepChoice(int *dev_csp, unsigned int *dev_poi, double *dev_skels, double *dev_us, double *dev_parms, double *dev_psio, double *dev_endpts, int *dev_keep)
{
  int bid=blockIdx.x;
   int size, start;
   double phitilde;
   dev_keep[bid]=1;
   dev_endpts[bid]=dev_skels[dev_csp[bid+1]-1+bid];
   size=dev_poi[bid];
   start=dev_csp[bid];
   if(exp(dev_parms[0]*dev_endpts[bid]-max(0.0,dev_parms[0])) < dev_us[bid])
    dev_keep[bid]=0;
   for (int j=0; j<size-1; j++)
   {
    phitilde=dev_parms[0]/2.0*(-dev_parms[0]*pow(dev_skels[start+j],2)+dev_skels[start+j]*(dev_parms[0]-2.0*dev_parms[1])+dev_parms[1]);
    if (phitilde > dev_psio[start+j])
     dev_keep[bid]=0;
   }
}

int main( void ) 
{ 
 /*declare objects with fixed size*/
 double s, p, phi, samp[SAMPSIZE], params[3], endpts[N];
 int y, z, i, j, k, poisum, cspoi[N+1], keep[N], size, start, index, kept, loops;
 unsigned int poi[N];

 /*set up random number generators*/
 hiprandGenerator_t gen; 
 hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32); 
 hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL); 
 const gsl_rng_type * rngT;
 gsl_rng_env_setup();
 rngT = gsl_rng_default;

 /*set parameters*/
 double T=0.1;
 double theta=0.00014;
 double ps[99];
 for (i=0; i<99; i++) 
 {
  ps[i]=((double)i+1.0)/99.0; 
 }
 double ss[59];
 for (i=0; i<1; i++) 
 {
  ss[i]=((double)i/2.0)-12.0; 
 }

 for (y=0; y<59; y++)
 {
  s=ss[y];
  phi=phiplus(s,theta);

  FILE *file3;    
  file3 = fopen("current_sb.txt","w");
   fprintf(file3, "%.1f\n", s);  
  fclose(file3); 

  for (z=0; z<99; z++)
  { 
   p=ps[z]; 

   FILE *file4;    
   file4 = fopen("current_pb.txt","w");
    fprintf(file4, "%.2f\n", p);  
   fclose(file4); 

   params[0]=s;
   params[1]=theta;
   params[2]=T;

   kept=0;
   loops=0;

   while (kept<SAMPSIZE)
   {

    //printf("%d\n",loops);
    //printf("%d\n",kept);
    
    /***sample from poisson point process***/
    //generate poisson sample
    unsigned int *dev_poi; 
     hipMalloc( (void**)&dev_poi, N*sizeof(int) ); 
    hiprandGeneratePoisson(gen, dev_poi, N, phi*T); 
    hipMemcpy(poi, dev_poi, N*sizeof(int), hipMemcpyDeviceToHost);  

    //compute sum of poisson sample
    poisum=0;
    for (j=0; j<N; j++)
    {
     poisum=poisum+poi[j];
    }

    //generate uniform samples for ts and psis
    double *ts, *psis;
     ts = (double *) malloc(poisum*sizeof(double));
     psis = (double *) malloc(poisum*sizeof(double));
    double *dev_ts, *dev_psis;
     hipMalloc( (void**)&dev_ts, poisum*sizeof(double) ); 
     hipMalloc( (void**)&dev_psis, poisum*sizeof(double) );
    hiprandGenerateUniformDouble(gen, dev_ts, poisum); 
    hipMemcpy(ts, dev_ts, poisum*sizeof(double), hipMemcpyDeviceToHost); 
    hipFree(dev_ts);  
    for (j=0; j<poisum; j++)
    {
     ts[j]=ts[j]*T;
    } 
    hiprandGenerateUniformDouble(gen, dev_psis, poisum); 
    hipMemcpy(psis, dev_psis, poisum*sizeof(double), hipMemcpyDeviceToHost); 
    hipFree(dev_psis);  
    for (j=0; j<poisum; j++)
    {  
     psis[j]=psis[j]*phi; 
    }

    /***sort each of N sets of t and reorder N sets of psi accordingly***/
    //compute cumulative sum for indexing sets
    cspoi[0]=0;
    for (j=0; j<N; j++)
    {
     cspoi[j+1]=cspoi[j]+poi[j];
    }

    //use quicksort to sort ts and reorder psis 
    double *psis_ord;
     psis_ord = (double *) malloc(poisum*sizeof(double));  
    for (k=0; k<N; k++)
    { 
     size=poi[k];
     start=cspoi[k];
     if (size==1)
     {
      psis_ord[start]=psis[start];
     }
     if (size>1)
     {
      double *a;
       a = (double *) malloc(size*sizeof(double));
      int *o;
       o = (int *) malloc(size*sizeof(int));
      for (j=0; j<size; j++)
      {
       a[j]=ts[start+j];
       o[j]=j;
      }
      quick_sort(a,o,0,size-1);
      for (j=0; j<size; j++)
      {
       ts[start+j]=a[j];
       psis_ord[start+j]=psis[start+o[j]];
      }
      free(a);
      free(o);
     }
    }
    free(psis);

    /***compute differences among consecutive ts within each set***/
    double *tdiffs;
     tdiffs = (double *) malloc((N+poisum)*sizeof(double));
    double *dev_tdiffs, *dev_parms, *dev_tss;
     hipMalloc( (void**)&dev_tdiffs, (N+poisum)*sizeof(double) );
     hipMalloc( (void**)&dev_parms, 3*sizeof(double) );
     hipMalloc( (void**)&dev_tss, poisum*sizeof(double) );
    int *dev_csp;
     hipMalloc( (void**)&dev_csp, (N+1)*sizeof(int) );
    hipMemcpy( dev_tss, ts, poisum*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( dev_csp, cspoi, (N+1)*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_parms, params, 3*sizeof(double), hipMemcpyHostToDevice );
    TimeDiffs<<<N,1>>>(dev_csp,dev_poi,dev_parms,dev_tss,dev_tdiffs);
    hipMemcpy( tdiffs, dev_tdiffs, (N+poisum)*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dev_tss);
  
    /***compute parameters for normal approximation***/
    double *naMeans, *naVars;
     naMeans = (double *) malloc((N+poisum)*sizeof(double));
     naVars = (double *) malloc((N+poisum)*sizeof(double));
    double *dev_means, *dev_vars;
     hipMalloc( (void**)&dev_means, (N+poisum)*sizeof(double) );
     hipMalloc( (void**)&dev_vars, (N+poisum)*sizeof(double) );
    NormApproxParams<<<N,1>>>(dev_csp,dev_poi,dev_tdiffs,dev_parms,dev_means,dev_vars);
    hipMemcpy(naMeans, dev_means, (N+poisum)*sizeof(double), hipMemcpyDeviceToHost); 
    hipMemcpy(naVars, dev_vars, (N+poisum)*sizeof(double), hipMemcpyDeviceToHost);     
    hipFree(dev_tdiffs);
    hipFree(dev_means);
    hipFree(dev_vars);
  
    /***generate draws from neutral wright-fisher process***/
    double *ms, *skels;
     ms = (double *) malloc((N+poisum)*sizeof(double));
     skels = (double *) malloc((N+poisum)*sizeof(double));
    long long *rms;
     rms = (long long *) malloc((N+poisum)*sizeof(long long));
    int *ls;
     ls = (int *) malloc((N+poisum)*sizeof(int));
    gsl_rng * r;
     r = gsl_rng_alloc (rngT);
    index=0;
    for (j=0; j<N+poisum; j++) 
    {
     if (tdiffs[j]>0.000001)
     {
      ms[j]=gsl_ran_gaussian(r,sqrt(naVars[j]));
      ms[j]=ms[j]+naMeans[j];
     }
     else
      ms[j]=(2.0*theta-1.0)/(exp(0.5*(2.0*theta-1.0)*0.000001)-1.0);
     rms[j]=round(ms[j]);
     if (j==cspoi[index]+index)
     {
      ls[j]=gsl_ran_binomial(r,p,rms[j]);
      index++;
     }
     else
      ls[j]=gsl_ran_binomial(r,skels[j-1],rms[j]);
     skels[j]=gsl_ran_beta(r,theta+(double)ls[j],theta+(double)rms[j]-(double)ls[j]);
    }
    gsl_rng_free(r);  
    free(naMeans);
    free(naVars);
    free(ms);
    free(rms);
    free(ls);

    /***choose whether to accept or reject each skeleton***/
    //generate random uniforms with which to make decisions
    double *dev_us;
     hipMalloc( (void**)&dev_us, N*sizeof(double) ); 
    hiprandGenerateUniformDouble(gen, dev_us, N); 
  
    //identify endpoints of skeletons and which ones to accept
    int *dev_keep;
     hipMalloc( (void**)&dev_keep, N*sizeof(int) );
    double *dev_skels, *dev_psio, *dev_endpts;
     hipMalloc( (void**)&dev_skels, (N+poisum)*sizeof(double) );
     hipMalloc( (void**)&dev_psio, poisum*sizeof(double) );
     hipMalloc( (void**)&dev_endpts, N*sizeof(double) );
    hipMemcpy( dev_skels, skels, (N+poisum)*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( dev_psio, psis_ord, poisum*sizeof(double), hipMemcpyHostToDevice );
    KeepChoice<<<N,1>>>(dev_csp,dev_poi,dev_skels,dev_us,dev_parms,dev_psio,dev_endpts,dev_keep);
    hipMemcpy(endpts, dev_endpts, N*sizeof(double), hipMemcpyDeviceToHost); 
    hipMemcpy(keep, dev_keep, N*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_poi);
    hipFree(dev_csp);
    hipFree(dev_keep);
    hipFree(dev_skels);
    hipFree(dev_parms);
    hipFree(dev_psio);
    hipFree(dev_endpts);
    hipFree(dev_us);
    free(ts);
    free(psis_ord);
    free(tdiffs);
    free(skels);

    /***store accepted endpoints and keep track of number accepted***/
    for (j=0; j<N; j++)
    {
     if (keep[j]==1 && kept<SAMPSIZE)
     {  
      samp[kept]=endpts[j];
      kept++;
     }
    }
    loops++;
   }

   /***append final sample to output file***/
   FILE *file1;  
   file1 = fopen("final_samples_cuda.txt","a");
   for(j=0; j<SAMPSIZE; j++)
   { 
    fprintf(file1, "%.6f\n", samp[j]);  
   } 
   fclose(file1); 
  } 
 }  
 return 1; 
}
